#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

void printFloatList(float *buff, char* buffName, int buffLen){
    printf("WUICHAK: %s - ", buffName);
    for(int i = 0; i < buffLen; i++){
        printf("%.2f, ", buff[i]);
    }
    printf("\n");
}

int main(int argc, char* argv[])
{
  int nccl_version;
  NCCLCHECK(ncclGetVersion(&nccl_version));
  printf("NCCL VERSION - %d\n", nccl_version);
  //managing 4 devices
  int nDev = 4;
  int size = 10;
//   int devs[2] = { 0, 1};
  int devs[4] = { 0, 1, 2, 3};
  ncclComm_t comms[nDev];


  //allocating and initializing device buffers
  float** sendbuff = (float**)malloc(nDev * size * sizeof(float*));
  float** recvbuff = (float**)malloc(nDev * size * sizeof(float*));
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);


  float *databuff = new float[size] ();    
  float *hostbuff = (float*)malloc( size * sizeof(float));
  for (int i = 0; i < nDev; ++i) {
    for(int idatum = 0; idatum < size; idatum++){
        databuff[idatum] = (1+idatum)*(i+1) ;
    }
    printFloatList(databuff, "databuff", size);
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
    CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
    CUDACHECK(hipMemcpy(sendbuff[i], databuff, size * sizeof(float), hipMemcpyHostToDevice));
    // CUDACHECK(cudaMemset(sendbuff[i], 1, size * sizeof(float)));
    CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float)));
    CUDACHECK(hipStreamCreate(s+i));
  }


  //initializing NCCL
  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

   //calling NCCL communication API. Group API is required when using
   //multiple devices per thread
  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < nDev; ++i){
    int send_peer = (i + 1) % nDev;
    int recv_peer = (i + nDev - 1) % nDev;
    // NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum,
    //     comms[i], s[i]));
    NCCLCHECK(ncclSend((const void*)sendbuff[i], size, ncclFloat, send_peer, comms[i], s[i]));
    NCCLCHECK(ncclRecv((void*)recvbuff[i], size, ncclFloat, recv_peer, comms[i], s[i]));
    
    printf("WUICHAK: p2p nDev-%d\n", i);
  }
  NCCLCHECK(ncclGroupEnd());


  //synchronizing on CUDA streams to wait for completion of NCCL operation
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }


  //free device buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    printf("Device %d:\n", i);
    CUDACHECK(hipMemcpy(hostbuff, sendbuff[i], size * sizeof(float), hipMemcpyDeviceToHost));
    printFloatList(hostbuff, "sendbuff", size);
    CUDACHECK(hipMemcpy(hostbuff, recvbuff[i], size * sizeof(float), hipMemcpyDeviceToHost));
    printFloatList(hostbuff, "recvbuff", size);
    
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }


  //finalizing NCCL
  for(int i = 0; i < nDev; ++i)
      ncclCommDestroy(comms[i]);


  printf("Success \n");
  return 0;
}
